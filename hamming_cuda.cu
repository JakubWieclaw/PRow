#include "hip/hip_runtime.h"
#include "solver/hamming.h"
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

__device__ int count_hamming_distance_cuda(const bool* a, const bool* b, int m) {
    int distance = 0;
    for (int i = 0; i < m; i++) {
        if (a[i] != b[i]) {
            distance++;
        }
    }
    return distance;
}

__global__ void count_all_pairs_with_distance_equal_n_cuda(const bool* d_sequences, int* d_pairs, int num_sequences, int m, int n, int* d_count) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int total_pairs = num_sequences * (num_sequences - 1) / 2;

    if (idx < total_pairs) {
        int i = idx / (num_sequences - 1);
        int j = idx % (num_sequences - 1);
        if (j >= i) j++;

        const bool* a = d_sequences + i * m;
        const bool* b = d_sequences + j * m;

        if (count_hamming_distance_cuda(a, b, m) == n) {
            int index = atomicAdd(d_count, 1);
            d_pairs[index * 2] = i;
            d_pairs[index * 2 + 1] = j;
        }
    }
}

std::vector<std::pair<int, int>> cuda_count_all_pairs_with_distance_equal_n(const std::vector<std::vector<bool>>& vector_of_sequences, int n) {
    int num_sequences = vector_of_sequences.size();
    int m = vector_of_sequences[0].size();
    int total_pairs = num_sequences * (num_sequences - 1) / 2;

    bool* h_sequences = new bool[num_sequences * m];
    int* h_pairs = new int[total_pairs * 2];
    int h_count = 0;

    for (int i = 0; i < num_sequences; i++) {
        for (int j = 0; j < m; j++) {
            h_sequences[i * m + j] = vector_of_sequences[i][j];
        }
    }

    bool* d_sequences;
    int* d_pairs;
    int* d_count;

    hipMalloc(&d_sequences, num_sequences * m * sizeof(bool));
    hipMalloc(&d_pairs, total_pairs * 2 * sizeof(int));
    hipMalloc(&d_count, sizeof(int));

    hipMemcpy(d_sequences, h_sequences, num_sequences * m * sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(d_count, &h_count, sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (total_pairs + blockSize - 1) / blockSize;
    count_all_pairs_with_distance_equal_n_cuda<<<numBlocks, blockSize>>>(d_sequences, d_pairs, num_sequences, m, n, d_count);

    hipMemcpy(&h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_pairs, d_pairs, h_count * 2 * sizeof(int), hipMemcpyDeviceToHost);

    std::vector<std::pair<int, int>> pairs;
    for (int i = 0; i < h_count; i++) {
        pairs.push_back(std::make_pair(h_pairs[i * 2], h_pairs[i * 2 + 1]));
    }

    hipFree(d_sequences);
    hipFree(d_pairs);
    hipFree(d_count);
    delete[] h_sequences;
    delete[] h_pairs;

    return pairs;
}
